
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>

// Print array kernel.
__global__ void print_array(int *array, int size) {
  if(threadIdx.x == 0) {
    for(int i = 0; i < size; i++) {
      printf("%d ", array[i]);
    }
    printf("\n");
  }
}

// Print the sudoku kernel.
__global__ void printSudokuDeviceKernel(int *sudoku, int sqrtN)
{
  int n = sqrtN * sqrtN;
  printf(
      "#############################\nSUDOKU\n#############################\n");

  for (int i = 0; i < n; ++i)
  {
    if (i % sqrtN == 0)
    {
      printf("----------------------\n");
    }
    for (int j = 0; j < n; ++j)
    {
      if (j % sqrtN == 0)
      {
        printf("|");
      }
      if (sudoku[i * n + j] <= 15)
      {
        printf("%X ", sudoku[i * n + j]);
      }
      else
      {
        char c = 'F' + (sudoku[i * n + j] - 15);
        printf("%c ", c);
      }
    }
    printf("|\n");
    if (i == n - 1)
    {
      printf("----------------------\n\n");
    }
  }
}

// Print the sudoku on device.
__device__ void printSudokuDevice(int *sudoku, int sqrtN)
{
  int n = sqrtN * sqrtN;
  printf(
      "#############################\nSUDOKU\n#############################\n");

  for (int i = 0; i < n; ++i)
  {
    if (i % sqrtN == 0)
    {
      printf("----------------------\n");
    }
    for (int j = 0; j < n; ++j)
    {
      if (j % sqrtN == 0)
      {
        printf("|");
      }
      if (sudoku[i * n + j] <= 15)
      {
        printf("%X ", sudoku[i * n + j]);
      }
      else
      {
        char c = 'F' + (sudoku[i * n + j] - 15);
        printf("%c ", c);
      }
    }
    printf("|\n");
    if (i == n - 1)
    {
      printf("----------------------\n\n");
    }
  }
}

// Print the expanded sudoku.
void printExpandedSudoku(int *sudoku, int sqrtN, int expand)
{
  int n = sqrtN * sqrtN;
  printf("#############################\nEXPANDED SUDOKU\n#############################\n");
  for (int k = 0; k < expand; ++k)
  {
    printf(
        "#############################\nSUDOKU %d\n#############################\n", k);
    int currentSudoku = k * n * n;
    for (int i = 0; i < n; ++i)
    {
      if (i % sqrtN == 0)
      {
        printf("----------------------\n");
      }
      for (int j = 0; j < n; ++j)
      {
        if (j % sqrtN == 0)
        {
          printf("|");
        }
        if (sudoku[currentSudoku + i * n + j] <= 15)
        {
          printf("%X ", sudoku[currentSudoku + i * n + j]);
        }
        else
        {
          char c = 'F' + (sudoku[currentSudoku + i * n + j] - 15);
          printf("%c ", c);
        }
      }
      printf("|\n");
      if (i == n - 1)
      {
        printf("----------------------\n\n");
      }
    }
  }
}

// Print the expanded sudoku on device.
__global__ void printExpandedSudokuKernel(int *sudoku, int sqrtN, int expand)
{
  int n = sqrtN * sqrtN;
  printf("#############################\nEXPANDED SUDOKU\n#############################\n");
  for (int k = 0; k < expand; ++k)
  {
    printf(
        "#############################\nSUDOKU %d\n#############################\n", k);
    int currentSudoku = k * n * n;
    for (int i = 0; i < n; ++i)
    {
      if (i % sqrtN == 0)
      {
        printf("----------------------\n");
      }
      for (int j = 0; j < n; ++j)
      {
        if (j % sqrtN == 0)
        {
          printf("|");
        }
        if (sudoku[currentSudoku + i * n + j] <= 15)
        {
          printf("%X ", sudoku[currentSudoku + i * n + j]);
        }
        else
        {
          char c = 'F' + (sudoku[currentSudoku + i * n + j] - 15);
          printf("%c ", c);
        }
      }
      printf("|\n");
      if (i == n - 1)
      {
        printf("----------------------\n\n");
      }
    }
  }
}

void waitForErrors()
{
  do
  {
    sleep(1);
    printf("Looking for errors...\n");

    // size_t available, total;
    // cudaMemGetInfo(&available, &total);

    // int *buf_d = 0;
    // size_t nwords = total / sizeof(int);
    // size_t words_per_Mb = 1 << 20 / sizeof(int);
    // printf("Memory available: %zu | Memory total: %zu\n", available, total);

    hipError_t error = hipGetLastError();
    printf("Error: %s\n", hipGetErrorString(error));
    
    if (error != hipSuccess)
    {
      exit(1);
    }
  } while (true);
}